#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace {

CUDADEV_KERNEL_FW_X(log, ::logf(px[i]));
CUDADEV_KERNEL_BW_X(log, pgy[i] / px[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(log);
CUDADEV_BW_X(log);

}  // namespace devices
}  // namespace primitiv
