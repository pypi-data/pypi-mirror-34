#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace {

__global__ void set_const_dev(float k, std::uint32_t size, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = k;
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::reset_tensor_impl(float k, Tensor &x) {
  const std::uint32_t size = x.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(k, size, MDATA(x));
}

}  // namespace devices
}  // namespace primitiv
