#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace primitiv {
namespace devices {

void CUDA16::copy_tensor_impl(const Tensor &x, Tensor &y) {
  switch (x.device().type()) {
    case Device::DeviceType::NAIVE:
      reset_tensor_by_array(CDATA(float, x), y);
      break;
    //case Device::DeviceType::CUDA:
      // TODO(odashi): Implement this section.
    case Device::DeviceType::CUDA16:
      CUDA_CALL(::hipSetDevice(dev_id_));
      // NOTE(odashi):
      // If source/destination devices use the unified memory space on the 64
      // bits machine, we can perform ::hipMemcpy to copy data beyond devices.
      CUDA_CALL(::hipMemcpyAsync(
            MDATA(half, y), CDATA(half, x),
            sizeof(half) * x.shape().size(),
            hipMemcpyDeviceToDevice, 0));
      break;
    default:
      reset_tensor_by_vector(x.to_vector(), y);
  }
}

}  // namespace devices
}  // namespace primitiv
