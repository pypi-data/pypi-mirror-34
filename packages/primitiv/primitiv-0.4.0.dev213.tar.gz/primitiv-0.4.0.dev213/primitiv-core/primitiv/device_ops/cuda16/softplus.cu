#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16_KERNEL_FW_X(
    softplus, ::fmaxf(X_VAL, .0f) + ::logf(1.f + ::expf(-::fabs(X_VAL))));
CUDA16_KERNEL_BW_X(softplus, (.5f + .5f * ::tanhf(.5f * X_VAL)) * GY_VAL);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X(softplus);
CUDA16_DEV_BW_X(softplus);

}  // namespace devices
}  // namespace primitiv
