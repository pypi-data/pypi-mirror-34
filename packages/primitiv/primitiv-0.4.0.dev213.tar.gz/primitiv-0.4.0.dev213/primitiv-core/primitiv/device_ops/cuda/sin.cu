#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace {

CUDADEV_KERNEL_FW_X(sin, ::sinf(px[i]));
CUDADEV_KERNEL_BW_X(sin, ::cosf(px[i]) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(sin);
CUDADEV_BW_X(sin);

}  // namespace devices
}  // namespace primitiv
