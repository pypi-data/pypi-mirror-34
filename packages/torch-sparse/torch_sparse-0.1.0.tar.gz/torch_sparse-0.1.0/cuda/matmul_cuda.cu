#include <ATen/ATen.h>

#include <hipsparse.h>

#define CSRGEMM(TYPE, ...)                                                     \
  [&] {                                                                        \
    const at::Type &the_type = TYPE;                                           \
    switch (the_type.scalarType()) {                                           \
    case at::ScalarType::Float: {                                              \
      using scalar_t = float;                                                  \
      return hipsparseScsrgemm(__VA_ARGS__);                                    \
    }                                                                          \
    case at::ScalarType::Double: {                                             \
      using scalar_t = double;                                                 \
      return hipsparseDcsrgemm(__VA_ARGS__);                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '%s'", the_type.toString());               \
    }                                                                          \
  }()

static hipsparseHandle_t cusparse_handle = 0;

static void init_cusparse() {
  if (cusparse_handle == 0) {
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
  }
}

std::tuple<at::Tensor, at::Tensor> spspmm_cuda(at::Tensor A, at::Tensor B) {
  init_cusparse();

  A = A.coalesce();
  B = B.coalesce();

  auto m = A.size(0);
  auto n = B.size(1);
  auto k = A.size(1);

  auto nnzA = A._nnz();
  auto nnzB = B._nnz();

  auto valueA = A._values();
  auto indexA = A._indices().toType(at::kInt);
  auto row_ptrA = at::empty(indexA.type(), {m + 1});
  hipsparseXcoo2csr(cusparse_handle, indexA[0].data<int>(), nnzA, k,
                   row_ptrA.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);
  auto colA = indexA[1];
  hipMemcpy(row_ptrA.data<int>() + m, &nnzA, sizeof(int),
             hipMemcpyHostToDevice);

  auto valueB = B._values();
  auto indexB = B._indices().toType(at::kInt);
  auto row_ptrB = at::empty(indexB.type(), {k + 1});
  hipsparseXcoo2csr(cusparse_handle, indexB[0].data<int>(), nnzB, k,
                   row_ptrB.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);
  auto colB = indexB[1];
  hipMemcpy(row_ptrB.data<int>() + k, &nnzB, sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  int nnzC;
  auto row_ptrC = at::empty(indexA.type(), {m + 1});
  hipsparseXcsrgemmNnz(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnzA,
                      row_ptrA.data<int>(), colA.data<int>(), descr, nnzB,
                      row_ptrB.data<int>(), colB.data<int>(), descr,
                      row_ptrC.data<int>(), &nnzC);
  auto colC = at::empty(indexA.type(), {nnzC});
  auto valueC = at::empty(valueA.type(), {nnzC});

  CSRGEMM(valueC.type(), cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
          HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnzA,
          valueA.data<scalar_t>(), row_ptrA.data<int>(), colA.data<int>(),
          descr, nnzB, valueB.data<scalar_t>(), row_ptrB.data<int>(),
          colB.data<int>(), descr, valueC.data<scalar_t>(),
          row_ptrC.data<int>(), colC.data<int>());

  auto rowC = at::empty(indexA.type(), {nnzC});
  hipsparseXcsr2coo(cusparse_handle, row_ptrC.data<int>(), nnzC, m,
                   rowC.data<int>(), HIPSPARSE_INDEX_BASE_ZERO);

  auto indexC = at::stack({rowC, colC}, 0).toType(at::kLong);

  return std::make_tuple(indexC, valueC);
}
