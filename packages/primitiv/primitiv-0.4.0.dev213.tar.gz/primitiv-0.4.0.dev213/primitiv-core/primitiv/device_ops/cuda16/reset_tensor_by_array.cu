#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

__global__ void fp32to16(const float *src, half *dest, std::size_t size) {
  const std::size_t i = IDX;
  if (i < size) dest[i] = ::__float2half(src[i]);
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::reset_tensor_by_array_impl(const float values[], Tensor &x) {
  const std::size_t size = x.shape().size();
  const std::size_t gs = GRID_SIZE(size, dim1_x_);

  auto temp = state_->pool.allocate(sizeof(float) * size);
  float *temp_ptr = static_cast<float *>(temp.get());

  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        temp_ptr, values, sizeof(float) * size, hipMemcpyHostToDevice));
  ::fp32to16<<<gs, dim1_x_>>>(temp_ptr, MDATA(half, x), size);
}

}  // namespace devices
}  // namespace primitiv
