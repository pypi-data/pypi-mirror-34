#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

__global__ void fp32to16(const float *px, std::size_t size, half *py) {
  const std::size_t i = IDX;
  if (i < size) py[i] = ::__float2half(px[i]);
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::random_normal_impl(float mean, float sd, Tensor &y) {
  const std::size_t size = y.shape().size();
  const std::size_t gs = GRID_SIZE(size, dim1_x_);
  auto temp = state_->pool.allocate(size * sizeof(float));
  float *temp_ptr = static_cast<float *>(temp.get());

  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateNormal(
        state_->hiprand.get(), temp_ptr, size, mean, sd));
  ::fp32to16<<<gs, dim1_x_>>>(temp_ptr, size, MDATA(half, y));
}

}  // namespace devices
}  // namespace primitiv
