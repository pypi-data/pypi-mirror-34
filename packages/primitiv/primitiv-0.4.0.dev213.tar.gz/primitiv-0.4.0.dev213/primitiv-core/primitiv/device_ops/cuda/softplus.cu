#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace {

CUDADEV_KERNEL_FW_X(
    softplus, ::fmaxf(px[i], .0f) + ::logf(1.f + ::expf(-::fabs(px[i]))));
CUDADEV_KERNEL_BW_X(softplus, (.5f + .5f * ::tanhf(.5f * px[i])) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(softplus);
CUDADEV_BW_X(softplus);

}  // namespace devices
}  // namespace primitiv
