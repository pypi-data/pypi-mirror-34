#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda/common.h>

namespace primitiv {
namespace devices {

std::vector<float> CUDA::tensor_to_vector_impl(const Tensor &x) {
  const std::uint32_t size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        ret.data(), CDATA(x), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

}  // namespace devices
}  // namespace primitiv
