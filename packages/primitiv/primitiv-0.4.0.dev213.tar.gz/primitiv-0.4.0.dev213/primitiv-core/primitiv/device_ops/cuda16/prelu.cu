#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16_KERNEL_FW_X_CONST(prelu, ::fmaxf(X_VAL, .0f) + k * ::fminf(X_VAL, .0f));
CUDA16_KERNEL_BW_X_CONST(prelu, GY_VAL * ((X_VAL > .0f) + k * (X_VAL <= .0f)));

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X_CONST(prelu);
CUDA16_DEV_BW_X_CONST(prelu);

}  // namespace devices
}  // namespace primitiv
