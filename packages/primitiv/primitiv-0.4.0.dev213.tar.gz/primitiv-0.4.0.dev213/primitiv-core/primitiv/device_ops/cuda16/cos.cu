#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/internal/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16_KERNEL_FW_X(cos, ::cosf(X_VAL));
CUDA16_KERNEL_BW_X(cos, -::sinf(X_VAL) * GY_VAL);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X(cos);
CUDA16_DEV_BW_X(cos);

}  // namespace devices
}  // namespace primitiv
